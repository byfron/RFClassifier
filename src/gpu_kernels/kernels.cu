#include "hip/hip_runtime.h"
#define TILE_SIZE 32

__kernel void predict(__global uint32_t * left_child_arr,
		 	  		  __global uint32_t * right_child_arr,
					  __global int16_t * offset1_x_arr,
					  __global int16_t * offset1_y_arr,
  					  __global int16_t * offset2_x_arr,
					  __global int16_t * offset2_y_arr,
					  __global float * theshold_arr,
					  __global uint8_t * is_unary_arr,
					  __global uint8_t * label_arr,
					  __read_only image2d_t input_image,
					  __write_only image2d_t output_image)
{

	// const sampler_t sampler=CLK_NORMALIZED_COORDS_FALSE | CLK_ADDRESS_CLAMP | CLK_FILTER_NEAREST;
	// int2 pixelcoord = (int2) (get_global_id(0), get_global_id(1));
	// int width = get_image_width(input_image);
	// int height = get_image_height(input_image);

	// if (pixelcoord.x < width && pixelcoord.y < height) {

	// 	float z = read_imagef(input_image, sampler, (int2)(pixelcoord.x, pixelcoord.y));

 	// 	// check that the pixel is foreground (10 meters is background)
	// 	if (z == 10.0f) return;
	// 	int idx = 1;
		
	// 	while(true) {		    

	// 		if (idx == 0) { // is leaf node
	// 		   	uint8_t label = label_arr[idx];
	// 			write_imagef(output_image, pixelcoord, label);
	// 			return;
	// 		}
			
	// 	    // evaluate feature
	// 		///////////////////////////////////////////////////
	// 		float value;
	// 		int u = pixelcoord.x + int(offset1_x_arr[idx]/z);
	// 		int v = pixelcoord.y + int(offset1_y_arr[idx]/z);	

	// 		if (u >= width || v >= height) {
	// 		    value = 10.0;
	// 		}
	// 		else {
	// 			value = read_imagef(input_image, sampler, (int2)(u, v));
	// 		}

	// 		if (!is_unary_arr[idx]) {
	// 			float value2;	
	// 		    u = pixelcoord.x + int(offset2_x_arr[idx]/z);
	// 		    v = pixelcoord.y + int(offset2_y_arr[idx]/z);	
	// 			if (u >= width || v >= height) {
	// 		       z = 10.0;
	// 			}
	// 			else {
	// 			   z = read_imagef(input_image, sampler, (int2)(u, v));				   
	// 		    }
	// 		}

	// 		value -= z;
	// 		//////////////////////////////////////////////////

	// 		if (value < theshold_arr[idx]) { // falls to left
	// 		    idx = left_child_arr[idx];
	// 		}
	// 		else { // falls to right
	// 		    idx = right_child_arr[idx];
	// 		}
	// 	}		
	// }

}
